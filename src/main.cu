#include "hip/hip_runtime.h"
//
// Created by kango on 2023/06/17.
//
#include "Object/Body/Body.cuh"
#include "Object/Geometry/Sphere.cuh"
#include "Renderer/Camera/Camera.cuh"
#include "Renderer/Renderer.cuh"
#include "Renderer/Scene.cuh"
#include "vector"
#include <iostream>
#include <windows.h>

__global__ void cuda_hello() {
    printf("Hello World from GPU!\n");
}

void func1() {
    std::cout << "Hello, World!" << std::endl;

    cuda_hello<<<1, 1>>>();

    hipDeviceSynchronize();

    const Color offWhite(1.0, 0.97, 0.93);

    Sphere worldSphere(1e10, Eigen::Vector3d(0, 0, 0));
    const double room_r = 1e5;
    Sphere roomSpheres[] = {
            Sphere(room_r, Eigen::Vector3d{room_r - 15, 0, 0}),
            Sphere(room_r, Eigen::Vector3d{-(room_r - 15), 0, 0}),
            Sphere(room_r, Eigen::Vector3d{0, 0, room_r - 30}),
            Sphere(room_r, Eigen::Vector3d{0, room_r - 10, 0}),
            Sphere(room_r, Eigen::Vector3d{0, -(room_r - 10), 0})
    };
    Sphere lightSphere(1.0, Eigen::Vector3d(0, 8.9, 7));

    Sphere sphere(3.0, Eigen::Vector3d{0, -7, -6});
    Sphere sphere2(2.0, Eigen::Vector3d{-8, -8, -4});
    Sphere sphere3(2.0, Eigen::Vector3d{7, -8, -5});

    Body world(0.0, Material(M_DIFFUSE, Color::Zero(), 0.0), worldSphere);
    const double room_kd = 0.6;
    Body room[] = {
            Body(0.0, Material(M_DIFFUSE, Color(1.0, 0.01, 0.01), room_kd), roomSpheres[0]),
            Body(0.0, Material(M_DIFFUSE, Color(0.01, 1.0, 0.01), room_kd), roomSpheres[1]),
            Body(0.0, Material(M_DIFFUSE, offWhite, room_kd), roomSpheres[2]),
            Body(0.0, Material(M_DIFFUSE, offWhite, room_kd), roomSpheres[3]),
            Body(0.0, Material(M_DIFFUSE, offWhite, room_kd), roomSpheres[4]),
    };
    Body light(1000.0, Material(M_ZERO, Color(1, 1, 1)), lightSphere);
    Body body(0.0, Material(Color(0.3, 0.92, 0.95), 1.0, 0.0, 0.0), sphere);
    Body body2(0.0, Material(Color(0.6, 0.7, 0.5), 0.01, 0.9, 0.0), sphere2);
    Body body3(0.0, Material(codeToColor("#5900ff"), 0.01, 0.1, 0.8), sphere3);
    Body body4(0.0, Material(Color(0.76, 0.67, 1.0), 0.6, 0.3, 0.001), Sphere(1.5, Eigen::Vector3d{3, -8.5, 0}));
    Body body5(0.0, Material(codeToColor("#FFC800"), 0.8, 0.1, 0.001), Sphere(1.5, Eigen::Vector3d{-4, -8.5, -1}));
    Body body6(0.0, Material(codeToColor("#bdd458"), 0.3, 0.7, 0.0), Sphere(2.0, Eigen::Vector3d{-5.0, -8.0, -8.0}));
    Body body7(0.0, Material(codeToColor("#FFFF88"), 0.3, 0.1, 0.6), Sphere(2.0, Eigen::Vector3d{-0.5, -10.0, 5.0}));
    std::vector<Body> bodies{world, light, body, body2, body3, body4, body5, body6, body7};
    for(auto & i : room) {
        bodies.push_back(i);
    }

    const Eigen::Vector3d camOrg{0, -5, 15.0};
    const Camera camera(
            camOrg,
            body.getSphere().center - camOrg,
            1080, 16.0 / 9.0, 40, 1.0, (body.getSphere().center - camOrg).norm(), 1.5, 20
    );

    Scene scene(bodies.size(), camera, bodies.data(), Color::Zero());

    scene.printSceneInfo();

    LARGE_INTEGER freq;
    QueryPerformanceFrequency(&freq);

    hipDeviceSynchronize();

    LARGE_INTEGER start, end;
    QueryPerformanceCounter(&start);

    auto image = generateImageWithGPU(scene, 1e4).apply_reinhard_extended_tone_mapping().apply_bilateral_filter(4, 150, 10).apply_gamma_correction();

    QueryPerformanceCounter(&end);

    const double time = static_cast<double>(end.QuadPart - start.QuadPart) / freq.QuadPart;
    std::cout << "Generation time\t" << time << " [sec]" << std::endl;

    image.show();
    image.save("../../Results/sample.png");

    hipDeviceReset();
}

int main() {
    func1();
    return 0;
}
