#include "hip/hip_runtime.h"
//
// Created by kango on 2023/06/17.
//
#include "Object/Body/Body.cuh"
#include "Object/Geometry/Sphere.cuh"
#include "Renderer/Camera/Camera.cuh"
#include "Renderer/Renderer.cuh"
#include "Renderer/Scene.cuh"
#include "vector"
#include <iostream>
#include <windows.h>

__global__ void cuda_hello() {
    printf("Hello World from GPU!\n");
}

void func1() {
    std::cout << "Hello, World!" << std::endl;

    cuda_hello<<<1, 1>>>();

    hipDeviceSynchronize();

    const Color offWhite(1.0, 0.97, 0.93);

    Sphere worldSphere(1e10, Eigen::Vector3d(0, 0, 0));
    const double room_r = 1e5;
    Sphere roomSpheres[] = {
            Sphere(room_r, Eigen::Vector3d{room_r - 15, 0, 0}),
            Sphere(room_r, Eigen::Vector3d{-(room_r - 15), 0, 0}),
            Sphere(room_r, Eigen::Vector3d{0, 0, room_r - 30}),
            Sphere(room_r, Eigen::Vector3d{0, room_r - 10, 0}),
            Sphere(room_r, Eigen::Vector3d{0, -(room_r - 10), 0})
    };
    Sphere lightSphere(1.0, Eigen::Vector3d(0, 8.9, 7));

    Sphere sphere(3.0, Eigen::Vector3d{0, -7, -6});
    Sphere sphere2(2.0, Eigen::Vector3d{-8, -8, -4});
    Sphere sphere3(2.0, Eigen::Vector3d{7, -8, -5});

    Body world(0.0, Material(M_DIFFUSE, Color::Zero(), 0.0), worldSphere);
    const double room_kd = 0.6;
    Body room[] = {
            Body(0.0, Material(M_DIFFUSE, Color(1.0, 0.01, 0.01), room_kd), roomSpheres[0]),
            Body(0.0, Material(M_DIFFUSE, Color(0.01, 1.0, 0.01), room_kd), roomSpheres[1]),
            Body(0.0, Material(M_DIFFUSE, offWhite, room_kd), roomSpheres[2]),
            Body(0.0, Material(M_DIFFUSE, offWhite, room_kd), roomSpheres[3]),
            Body(0.0, Material(M_DIFFUSE, offWhite, room_kd), roomSpheres[4]),
    };
    Body light(1000.0, Material(M_ZERO, Color(1, 1, 1)), lightSphere);
    Body body(0.0, Material(Color(0.3, 0.92, 0.95), 1.0, 0.0, 0.0), sphere);
    Body body2(0.0, Material(Color(0.6, 0.7, 0.5), 0.01, 0.9, 0.0), sphere2);
    Body body3(0.0, Material(codeToColor("#5900ff"), 0.01, 0.1, 0.8), sphere3);
    Body body4(0.0, Material(Color(0.76, 0.67, 1.0), 0.6, 0.3, 0.001), Sphere(1.5, Eigen::Vector3d{3, -8.5, 0}));
    Body body5(0.0, Material(codeToColor("#FFC800"), 0.8, 0.1, 0.001), Sphere(1.5, Eigen::Vector3d{-4, -8.5, -1}));
    Body body6(0.0, Material(codeToColor("#bdd458"), 0.3, 0.7, 0.0), Sphere(2.0, Eigen::Vector3d{-5.0, -8.0, -8.0}));
    Body body7(0.0, Material(codeToColor("#FFFF88"), 0.3, 0.1, 0.6), Sphere(2.0, Eigen::Vector3d{-0.5, -10.0, 5.0}));
    std::vector<Body> bodies{world, light, body, body2, body3, body4, body5, body6, body7};
    for(auto & i : room) {
        bodies.push_back(i);
    }

    const Eigen::Vector3d camOrg{0, -5, 15.0};
    const Camera camera(
            camOrg,
            body.getSphere().center - camOrg,
            1080, 16.0 / 9.0, 40, 1.0, (body.getSphere().center - camOrg).norm(), 1.5, 20
    );

    Scene scene(bodies.size(), camera, bodies.data(), Color::Zero());

    scene.printSceneInfo();

    LARGE_INTEGER freq;
    QueryPerformanceFrequency(&freq);

    hipDeviceSynchronize();

    LARGE_INTEGER start, end;
    QueryPerformanceCounter(&start);

    auto image = generateImageWithGPU(scene, 1e4).apply_reinhard_extended_tone_mapping().apply_bilateral_filter(5, 150, 10).apply_gamma_correction();

    QueryPerformanceCounter(&end);

    const double time = static_cast<double>(end.QuadPart - start.QuadPart) / freq.QuadPart;
    std::cout << "Generation time\t" << time << " [sec]" << std::endl;

    image.show();
    image.save("../../Results/sample.png");

    hipDeviceReset();
}

void func2() {
    const auto floor_color = codeToColor("#f9c89b");
    const double room_r = 1e5;
    const double room_kd = 0.8;
    Sphere roomSpheres[] = {
            Sphere(room_r, Eigen::Vector3d{room_r - 40, 0, 0}),
            Sphere(room_r, Eigen::Vector3d{-(room_r - 40), 0, 0}),
            Sphere(room_r, Eigen::Vector3d{0, 0, room_r - 100}),
            Sphere(room_r, Eigen::Vector3d{0, room_r - 50, 0}),
            Sphere(room_r, Eigen::Vector3d{0, -(room_r - 50), 0})
    };

    Body room[] = {
            Body(0.0, Material(M_DIFFUSE, codeToColor("#2f5d50"), room_kd), roomSpheres[0]),
            Body(0.0, Material(M_DIFFUSE, codeToColor("#00a3af"), room_kd), roomSpheres[1]),
            Body(0.0, Material(M_DIFFUSE, floor_color, room_kd), roomSpheres[2]),
            Body(0.0, Material(M_DIFFUSE, floor_color, room_kd), roomSpheres[3]),
            Body(0.0, Material(M_DIFFUSE, floor_color, room_kd), roomSpheres[4]),
    };

    std::vector<Body> bodies {
            Body(25, Material(codeToColor("#e597b2"), 1.0, 0.0, 0.0), Sphere(10, Eigen::Vector3d(0, 40, 0))),
            Body(0.0, Material(Color(0.75, 0.25, 0.25), 0.8, 0.0, 0.0), Sphere(12, Eigen::Vector3d(15, -38, -30))),
            Body(0.0, Material(codeToColor("#e5e4e6"), 0.8, 0.0, 0.0), Sphere(15, Eigen::Vector3d(-18, -35, -45))),
            Body(0.0, Material(codeToColor("#b0c4de"), 0.3, 0.5, 0.0), Sphere(8, Eigen::Vector3d(-3, -42, 5))),
            Body(0.0, Material(codeToColor("#c75595"), 0.6, 0.01, 0.0), Sphere(8, Eigen::Vector3d(-15, -42, -10))),
            Body(0.0, Material(codeToColor("#bdb76b"), 0.3, 0.2, 0.2), Sphere(9, Eigen::Vector3d(12, -41, -3))),
    };

    for(int i = 0; i < 5; i++) {
        bodies.emplace_back(room[i]);
    }

    Eigen::Vector3d sum = Eigen::Vector3d::Zero();
    for(int i = 1; i < 6; ++i) sum += bodies[i].getSphere().center;
    const Eigen::Vector3d camOrg{0, 5, 100.0};
    const Eigen::Vector3d camdir = sum / 5.0 - camOrg;
    const Camera camera(
            camOrg,
            camdir,
            1920, 9.0 / 16.0, 50, 1.0, camdir.norm(), 0.5, 20
    );

    Scene scene(bodies.size(), camera, bodies.data(), Color::Zero());

    scene.printSceneInfo();

    LARGE_INTEGER freq;
    QueryPerformanceFrequency(&freq);

    hipDeviceSynchronize();

    LARGE_INTEGER start, end;
    QueryPerformanceCounter(&start);

    auto image = generateImageWithGPU(scene, 1e4).apply_reinhard_extended_tone_mapping().apply_bilateral_filter(5, 150, 10).apply_gamma_correction();

    QueryPerformanceCounter(&end);

    const double time = static_cast<double>(end.QuadPart - start.QuadPart) / freq.QuadPart;
    std::cout << "Generation time\t" << time << " [sec]" << std::endl;

    // image.show();
    image.save("../../Results/sample2.png");

    hipDeviceReset();
}

int main() {
    func2();
    return 0;
}
